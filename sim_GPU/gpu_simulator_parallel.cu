#include "hip/hip_runtime.h"
#include <memory>
#include <vector>
#include <iostream>
#include <fstream>
#include <cstring>
#include <cassert>
#include <cmath>
#include <sys/time.h>
#include <omp.h>
#include "trt.cuh"
//#include <torch/script.h> // One-stop header.
#include "wtime.h"
using namespace std;
#define NO_MEAN
#define GPU
typedef long unsigned Tick;
typedef long unsigned Addr;
#define TD_SIZE 51
#define ROBSIZE 400
#include "herror.h"
#define MAXSRCREGNUM 8
#define MAXDSTREGNUM 6
#define TD_SIZE 51
#define INST_SIZE 62
#define CONTEXTSIZE 111
#define ROBSIZE 400
#define TICK_STEP 500.0
#define FETCH_BANDWIDTH 3
#define RETIRE_BANDWIDTH 4
#define ML_SIZE (TD_SIZE * CONTEXTSIZE)
#define MIN_COMP_LAT 6
#define WARP
#define ILINEC_BIT 33
#define IPAGEC_BIT 38
#define DADDRC_BIT 42
#define DLINEC_BIT 43
#define DPAGEC_BIT 47
#define PC 51
#define ISADDR 52
#define ADDR 53
#define ADDREND 54
#define IWALK0 55
#define IWALK1 56
#define IWALK2 57
#define DWALK0 58
#define DWALK1 59
#define DWALK2 60
#define COMPLETETICK 61
#define WARPSIZE 32
#define TRACE_DIM 51
#define AUX_TRACE_DIM 10
//#define Total_Trace 1024

Tick Num = 0;

float factor[TD_SIZE];
float mean[TD_SIZE];
float default_val[TD_SIZE];

struct params{
   bool is_empty;
   bool is_full;
   int saturated;
   Tick fetched;
   Tick completeTick;
   bool eof;
   int ROB_flag;
};

class Inst {
	public:
	float *train_data;
  Tick inTick;
  Tick completeTick;
  Tick tickNum;
  Tick trueFetchTick;
  Tick trueCompleteTick;
  int trueFetchClass;
  int trueCompleteClass;
  Addr pc;
  int isAddr;
  Addr addr;
  Addr addrEnd;
  Addr iwalkAddr[3];
  Addr dwalkAddr[3];
  int offset;
  //H_ERR(hipMalloc((void **)&train_data_d, sizeof(int)*TD_SIZE));
  // Read simulation data.



  __device__ void
dis(float *data, int size, int rows)
{
        for(int i=0;i<rows;i++)
        {
                for(int j=0; j<size;j++){
                printf("%.1f  ",data[i*size+j]);
                }
                printf("\n");
        }
}



  Inst(){} 
  Inst(float *pointer){
	train_data= pointer;
   }
  
 bool read_sim_mem(float *trace, uint64_t *aux_trace, float *train_d, int index) {
    train_data= train_d;
    trueFetchClass= trace[0];
    trueFetchTick= trace[1];
    trueCompleteClass= trace[2];
    trueCompleteTick= trace[3];
    pc= aux_trace[0];
    offset= INST_SIZE * index;
    //cout<< "Offset: "<< offset <<"   Memory: "<<train_data;
    //assert(trueCompleteTick >= MIN_COMP_LAT);
    for (int i = 4; i < TD_SIZE; i++) {
      train_data[i+offset]= trace[i]/factor[i];
      //cout<< trace[i]<<"\t" << train_data[i+offset]<<"\n";
    }
    train_data[0 + offset] = train_data[1 + offset] = 0.0;
    train_data[2 + offset] = train_data[3 + offset] = 0.0;
    //aux_trace >> isAddr >> addr >> addrEnd;
    isAddr= aux_trace[1];
    addr= aux_trace[2];
    addrEnd= aux_trace[3];
    for (int i = 0; i < 3; i++)
      iwalkAddr[i]=aux_trace[3+i];
    for (int i = 0; i < 3; i++)
      dwalkAddr[i]=aux_trace[6+i];
    train_data[PC]=(float)pc;
    train_data[ISADDR]=(float)isAddr;
    train_data[ADDR]=(float)addr;
    train_data[ADDREND]=(float)addrEnd;
    train_data[IWALK0]=(float)iwalkAddr[0];
    train_data[IWALK1]=(float)iwalkAddr[1];
    train_data[IWALK2]=(float)iwalkAddr[2];
    train_data[DWALK0]=(float)dwalkAddr[0];
    train_data[DWALK1]=(float)dwalkAddr[1];
    train_data[DWALK2]=(float)dwalkAddr[2];
    //cout << "in: ";
        for (int i = 0; i < TD_SIZE; i++)
        {}		      //cout << train_data[i] << " ";
	    //cout << "\n";
    return true;

 }	
};

class ROB{
public:
    float *insts;
    int head= 0;
    int tail= 0;
    int len= 0;
    bool saturated= false; 
    ~ROB(){};
    ROB(){
        H_ERR(hipMalloc((void **)&insts, sizeof(float)*(ROBSIZE*INST_SIZE)));
    };
    __host__ __device__ int inc(int input) {
	if (input == ROBSIZE)
          return 0;
        else
          return input + 1;
    }

    __host__ __device__ int dec(int input) {
	if (input == 0)
          return ROBSIZE;
        else
          return input - 1;
    }
    __host__ __device__ bool is_empty() { return head == tail; }
    __host__ __device__ bool is_full() { return head == inc(tail); }

__host__ __device__
     int add() {
    assert(!is_full());
    int old_tail = tail;
    tail = inc(tail);
    len+= 1;
    //printf("index updated.\n");
    return old_tail;
  }
    
    __device__
    int getHead() {
        return head;
      }

__device__ void
	retire(){
		assert(!is_empty());
		head= inc(head);
		len-=1;
	}

 __device__
 int retire_until(Tick tick, float *insts) {
	int retired = 0;
	while (!is_empty() && insts[COMPLETETICK] <= tick) {
		retire();
		retired++;
	}
	return retired;
 }


  __device__ void
dis(float *data, int size, int rows)
{
        for(int i=0;i<rows;i++)
        {
                for(int j=0; j<size;j++){
                printf("%.1f  ",data[i*size+j]);
                }
                printf("\n");
        }
}



	  __device__
    void update_fetch_cycle(Tick tick, Tick curTick, float *factor, float *insts) {
        int TID= (blockIdx.x * blockDim.x) + threadIdx.x;
	//int warpID= TID / WARPSIZE;
	int  warpTID= threadIdx.x % WARPSIZE;
    	assert(!is_empty());
        int context;
	int start_context = dec(dec(tail));
        int end_context= dec(head);
	int length= len - 1;        
        int i= warpTID;
	//{printf("TID: %d, Index: %d,len: %d, Update: start: %d, end: %d\n",warpTID,i,len,start_context,end_context);}
	//for (int i = dec(dec(tail)); i != dec(head); i = dec(i)) {
      if(warpTID==0){
        printf("ROB:, head: %d, tail: %d \n", head, tail);
        dis(insts, INST_SIZE, 4);
       }  
	__syncwarp();
	while(i<length){
          //printf("I: %d\n",i);
		  context = start_context -i;
		  context= (context>=0)?context:context+ROBSIZE;
		  float *inst= insts + context * INST_SIZE;
		  printf("warpTID:%d, Context: %d, curTick: %ld, %.2f\n",warpTID,context, curTick, inst[COMPLETETICK]);
		 if (inst[COMPLETETICK] <= (float)curTick)
			{printf("COntext: %d, warpTID: %d, Curtick: %ld, Inst: %.2f,continue\n",context, warpTID, curTick,inst[COMPLETETICK]);i+=WARPSIZE;continue;}
        printf("Context: %d, Before, %.3f, %.3f, Next: %d\n",context, inst[0],inst[1],dec(i-32));  
	inst[0] += tick / factor[0];
          if (inst[0] >= 9 / factor[0])
            inst[0] = 9 / factor[0];
          inst[1] += tick / factor[1];
	  printf("Context: %d, After, %.3f, %.3f,Next: %d\n", context, inst[0],inst[1], dec(i-32));
          assert(inst[0] >= 0.0);
          assert(inst[1] >= 0.0);
          i+=WARPSIZE; 
	  }
	  __syncwarp();
      }

      
__device__ 
	  int make_input_data(float *context, float *insts, Tick tick, float *factor, float *default_val) {
 	//if(){printf("Here. Head: %d, Tail: %d\n",head,tail);}

 	int TID= (blockIdx.x * blockDim.x) + threadIdx.x;
	int warpID= TID / WARPSIZE;
	int  warpTID= TID % WARPSIZE;
 	int curr= dec(tail);
	int start_context= dec(dec(tail));
	int end_context= dec(head);
	assert(!is_empty());
        saturated = false;
	__shared__ int num[4];
        Addr pc = insts[curr * INST_SIZE + PC];
        int isAddr= insts[curr * INST_SIZE + ISADDR];
        Addr addr = insts[curr * INST_SIZE + ADDR];
        Addr addrEnd = insts[curr * INST_SIZE + ADDREND];
        Addr iwalkAddr[3], dwalkAddr[3];
        int i= warpTID;
	int length= len - 1;
	//if (warpTID==0){
	while(i<3){
	//for (int i = 0; i < 3; i++) {
          iwalkAddr[i] = insts[curr*INST_SIZE + IWALK0 + i];
          dwalkAddr[i] = insts[curr*INST_SIZE + DWALK0 + i];
        i++;
	}
	__syncwarp();
	i= warpTID;
	while(i > length){
	      int context_ = start_context -i;
              context_= (context_>=0)?context_:context_+ROBSIZE;
	      float *inst= insts + context_ * INST_SIZE;		
	  printf("ThreadID: %d, inst id: %d\n",warpTID, i);
	  if (inst[COMPLETETICK] <= tick)
            continue;
          if (num[warpID] >= CONTEXTSIZE) {
            saturated = true;
            return 0;
          }
          // Update context instruction bits.
          inst[ILINEC_BIT] = inst[PC] == pc ? 1.0 / factor[ILINEC_BIT] : 0.0;
          int conflict = 0;
          for (int j = 0; j < 3; j++) {
            if (inst[j] != 0 && inst[j] == iwalkAddr[j])
              conflict++;}
          inst[IPAGEC_BIT] = (float)conflict / factor[IPAGEC_BIT];
          inst[DADDRC_BIT] = (isAddr && insts[ISADDR] && addrEnd >= inst[ADDR] && addr <= inst[ADDREND]) ? 1.0 / factor[DADDRC_BIT] : 0.0;
          inst[DLINEC_BIT] = (isAddr && inst[ISADDR] && (addr) == (inst[ADDR])) ? 1.0 / factor[DLINEC_BIT] : 0.0;
          conflict = 0;
          if (isAddr && inst[ISADDR])
            for (int j = 0; j < 3; j++) {
              if (inst[j] != 0 && inst[j] == dwalkAddr[j])
                conflict++;}
          inst[DPAGEC_BIT] = (float)conflict / factor[DPAGEC_BIT];
          //std::copy(insts[i].train_data, insts[i].train_data + TD_SIZE, context + num * TD_SIZE);
          //num++;
	  atomicAdd(&num[warpID],1);
        i-=WARPSIZE;
	}
	__syncwarp();
        i= warpTID;
        while (i<TD_SIZE){
                //for (int i = num; i < CONTEXTSIZE; i++) { //printf("thread: %d, i: %d\n",warpTID,i);
                //if(warpTID==0){printf("");}
                int j= curr;
                while(j!= end_context){
                        context[i+j*TD_SIZE]= insts[j*INST_SIZE+i];
                        //printf("Context: %d, index: %d,pos: %d, thread: %d, write: %.2f\n", j,i,i+j*TD_SIZE,warpTID, default_val[i]);
                        j=dec(j);}
        i+=WARPSIZE;}
	__syncwarp();

	//printf("Adding default values.\n");
	i= warpTID;
	while (i<TD_SIZE){
        	//for (int i = num; i < CONTEXTSIZE; i++) { //printf("thread: %d, i: %d\n",warpTID,i);
		//if(warpTID==0){printf("");}
		int j= 1;
		while(j< CONTEXTSIZE){
			context[i+j*TD_SIZE]= default_val[i];
			//printf("Context: %d, index: %d,pos: %d, thread: %d, write: %.2f\n", j,i,i+j*TD_SIZE,warpTID, default_val[i]);
			j++;}
	i+=WARPSIZE;}
	__syncwarp();
	return 0;
      }
};


class ROB_d {
   public:
	ROB *rob;
       ROB_d(int Total_Trace){
       		//ROB rob[Total_Trace]; 		
		H_ERR(hipMalloc((void **)&rob, sizeof(ROB)*(Total_Trace)));

       }	
};


__global__ void
preprocess(ROB_d *rob_d, float *insts,  float *factor, float *mean, float *default_val, float *inputPtr, float *train_data, Tick *curTick_d, Tick *lastFetchTick_d, int Total_Trace )
{
    
    int TID=(blockIdx.x * blockDim.x) + threadIdx.x ;
    int warpID= TID/WARPSIZE;
    int warpTID = TID%WARPSIZE;
    int TotalWarp = (gridDim.x * blockDim.x) / WARPSIZE;
    int index,Total;
    ROB *rob;
    float *rob_pointer; 
#ifdef WARP	
    index= warpID;
    Total= TotalWarp;
#else
    index= blockIdx.x;
    Total= gridDim.x;
#endif
     while(index<Total_Trace){
     	rob = &rob_d->rob[index];
    	Tick curTick= curTick_d[index];
	Tick lastFetchTick= lastFetchTick_d[index];
	//if(warpTID==0) { printf("Read: Warp: %d, assigned: %d, next: %d\n",warpID, index, index + Total);}
    //push new instruction to respective ROB but not latency
    //if(warpTID==0) { printf("Read: Warp: %d, assigned: %d, next: %d\n",warpID, index, index + Total);}
    //int tail= rob->dec(tail);
    rob_pointer= insts + ROBSIZE * INST_SIZE * index;	
     float *input_Ptr = inputPtr + ML_SIZE * index;
    int i= warpTID+4; 
    while(i<INST_SIZE)
    {
	    rob_pointer[i+INST_SIZE * rob->tail]= train_data[i + warpID * INST_SIZE];
	    //printf("t: %d, i: %d, offset: %d\n",TID,i,train_offset);	
	    i+=WARPSIZE;		
    }
    __syncwarp();
    //if(warpTID==0) { printf("Inpt: %d\n",warpID);} 
    if(warpTID==0){
       	if(rob->is_full()){
	     printf("retired\n");
	    int retired = rob->retire_until(curTick, insts); }	
    	rob->add();
    //printf("Tail: %d, Curtick: %ld, lastFetchTick: %ld\n", rob->tail, curTick, lastFetchTick);
    }
	//printf("Update: ROB: %d, thread: %d, head:%d, tail: %d, newIndex: %d\n", index, threadIdx.x, rob->head, rob->tail, (index + gridDim.x * blockDim.x));
    __syncwarp();	
    //printf("Curtick: %ld, lastFetchTick: %ld\n", curTick, lastFetchTick);
    if (curTick != lastFetchTick) {
	    //if(warpTID==0){printf("update fetch\n");}
        	rob->update_fetch_cycle(curTick - lastFetchTick, curTick, factor, rob_pointer);
   	}
    __syncwarp();
    //if(TID==0){printf("update completed\n"); }
    //rob = &rob_d->rob[index]; 
    //while(index<Total_Trace){
	//if(warpTID==0) { printf("Make input: Warp: %d, assigned: %d,offset: %d, next: %d\n",warpID, index,ML_SIZE*index, index + Total);}
    rob->make_input_data(input_Ptr, rob_pointer, curTick, factor, default_val);          
    if(warpTID==0){
	//printf("Input_Ptr\n");
	//dis(input_Ptr, TD_SIZE, 4);
       }
    __syncwarp();
    index+= Total;    
    }
}

__device__ Tick
max(float x, Tick y){
   if(x>y){return x;}
   else{return y;}
}

__global__ void
result(Tick *curTick, int Total_Trace)
{
	Tick sum=0;
	for(int i=0;i<Total_Trace;i++)
	{
		sum+=curTick[i];
	}
	printf("Total CurTick: %ld\n",sum);
}

__global__ void
update( ROB_d *rob_d, float* output, float* insts, float* factor, float* mean, Tick *curTick, Tick *lastFetchTick, int Total_Trace ){
	      //float output[]={ 2.1987 ,0.4428,  0.0245 , 0.2029, 0.0094 , 0.1621};      
	      //printf("Here\n");
	      int TID=(blockIdx.x * blockDim.x) + threadIdx.x ;
      	      int index= TID;
	      ROB *rob;
	      while(index<Total_Trace){
	      int offset= index *2;
	      //printf("index: %d, thread: %d, offset: %d \n",index,TID,offset);
	      Tick nextFetchTick=0;
	      rob = &rob_d->rob[index];
	      int rob_offset= ROBSIZE * INST_SIZE * index; 
              int context_offset= rob->dec(rob->tail) * INST_SIZE;
	      float *rob_pointer = insts + rob_offset + context_offset;
	     printf("Head: %d, Tail: %d\n",rob->head, rob->tail); 
	     //printf("Index: %d, offset: %d,Fetch: %.4f, Finish: %.4f\n ",index,rob->tail,output[offset+0],output[offset+1]);
	      float fetch_lat = output[offset+0] * factor[1] + mean[1];
	      float finish_lat = output[offset+1] * factor[3] + mean[3];
	      int int_fetch_lat = round(fetch_lat);
	      int int_finish_lat = round(finish_lat);
	      if (int_fetch_lat < 0)
		int_fetch_lat = 0;
	      if (int_finish_lat < MIN_COMP_LAT)
		int_finish_lat = MIN_COMP_LAT; 
    	     rob_pointer[0]= (-int_fetch_lat - mean[0]) / factor[0];
 	     rob_pointer[1]= (-int_fetch_lat - mean[1]) / factor[1];
	     rob_pointer[2]= (int_finish_lat - MIN_COMP_LAT - mean[2]) / factor[2];
	     if (rob_pointer[2] >= 9 / factor[2])
	     	{rob_pointer[2] = 9 / factor[2];}
	     rob_pointer[3] = (int_finish_lat - mean[3]) / factor[3]; 
	      printf("Index: %d, offset: %d, Fetch: %.4f, Finish: %.4f, Rob0: %.2f, Rob1: %.2f, Rob2: %.2f, Rob3: %.2f\n",index,rob->tail,output[offset+0],output[offset+1],rob_pointer[0],rob_pointer[1],rob_pointer[2],rob_pointer[3]);
	     rob_pointer[COMPLETETICK]= curTick[index] + int_finish_lat + int_fetch_lat;
	     lastFetchTick[index]= curTick[index];
	     if(int_fetch_lat){
		   nextFetchTick= curTick[index] + int_fetch_lat;}
			if((rob->is_full() || rob->saturated) && int_fetch_lat){
				curTick[index]= max(rob_pointer[COMPLETETICK], nextFetchTick);}
			else if(int_fetch_lat){
				curTick[index]= nextFetchTick;}
			else if(rob->saturated || rob->is_full()){
				curTick[index]= rob_pointer[COMPLETETICK];}
		
	    	//printf("curTick: %ld, completeTick: %.2f, nextfetchTick: %ld, lastFetchTick: %ld \n",curTick[index],rob_pointer[rob_offset+COMPLETETICK],nextFetchTick,lastFetchTick[index]); 
		index+= (gridDim.x*blockDim.x);	
	      }
	     }

void display(float *data, int size, int rows)
{
	for(int i=0;i<rows;i++){
		for(int j=0;j<size;j++){
			printf("%.2f\t",data[i*size+j]);
		}
		printf("\n");
	}
}

void display(unsigned long *data, int size, int rows)
{
	        for(int i=0;i<rows;i++){
			for(int j=0;j<size;j++){
				printf("%.f\t",(float)data[i*size+j]);
			}
			printf("\n");
		}
}



float *read_numbers(char *fname, int sz) {
  float *ret = new float[sz];
  ifstream in(fname);
  //printf("Trying to read from %s\n", fname);
  for(int i=0;i<sz;i++)
    in >> ret[i];
  return ret;
}

int read_trace_mem(char trace_file[], char aux_trace_file[], float *trace, Tick *aux_trace, int instructions)
{
  FILE *trace_f=fopen(trace_file,"rb");
  if(!trace_f){
	printf("Unable to read trace binary.");
	return 1;
	}
    int r=fread(trace,sizeof(float),TRACE_DIM*instructions,trace_f);
    printf("read :%d values for trace.\n",r);
    //display(trace,TRACE_DIM,2);

  FILE *aux_trace_f=fopen(aux_trace_file,"rb");
  if(!aux_trace_f){
        printf("Unable to aux_trace binary.");
        return 1;
        }
    int k=fread(aux_trace,sizeof(Tick),AUX_TRACE_DIM*instructions,aux_trace_f);  
    printf("read :%d values for aux_trace.\n",k);
    //display(aux_trace,AUX_TRACE_DIM,2);
    return true;
}

int main(int argc, char *argv[]) {
printf("args count: %d\n",argc);
#ifdef CLASSIFY
  if (argc != 8) {
    cerr << "Usage: ./simulator_q <trace> <aux trace> <lat module> <class module> <variances> <# inst> <Total trace>" << endl;
    return 0;
  }
#else
  if (argc != 7) {
    cerr << "Usage: ./simulator_q <trace> <aux trace> <lat module> <variances> <Total trace> <#Insts>" << endl;
#endif
    return 0;
  } 
  int arg_idx=4;
  float *varPtr = read_numbers(argv[arg_idx++], TD_SIZE);
  for (int i = 0; i < TD_SIZE; i++) {
#ifdef NO_MEAN
    mean[i] = -0.0;
#endif
    factor[i] = sqrtf(varPtr[i]);
    default_val[i] = -mean[i] / factor[i]; 
    //cout<<default_val[i]<<" ";  
  }
  //cout<<endl;
  int Total_Trace= atoi(argv[arg_idx++]);
  int Instructions= atoi(argv[arg_idx++]);  
  std::string model_path(argv[3]);
  TRTUniquePtr< nvinfer1::ICudaEngine > engine{nullptr};
  TRTUniquePtr< nvinfer1::IExecutionContext > context{nullptr};
  deseralizer(engine,context,model_path);
  std::vector<void*> buffers(engine->getNbBindings());
  std::vector<nvinfer1::Dims> input_dims;
  std::vector<nvinfer1::Dims> output_dims;
  for (size_t i = 0; i < engine->getNbBindings(); ++i){
    	auto binding_size = getSizeByDim(engine->getBindingDimensions(i)) * sizeof(float);
	//hipMalloc(&buffers[i], binding_size);
	if (engine->bindingIsInput(i)){
            input_dims.emplace_back(engine->getBindingDimensions(i));}
	else{output_dims.emplace_back(engine->getBindingDimensions(i));}}
	if (input_dims.empty() || output_dims.empty()){
	    std::cerr << "Expect at least one input and one output for network\n";
	    return -1;
   	 }
  float *trace;
  Tick *aux_trace;
  trace=(float*) malloc(TRACE_DIM*Instructions*sizeof(float));
  aux_trace=(Tick*) malloc(AUX_TRACE_DIM*Instructions*sizeof(Tick));
  read_trace_mem(argv[1],argv[2],trace,aux_trace,Instructions); 
  int Batch_size= Instructions / Total_Trace;
  cout << " Iterations: "<<Batch_size<<endl;
  //cout<<"Parameters read..\n";
   omp_set_num_threads(96);
   double measured_time = 0.0;
  ROB_d  *rob_d;
    Tick Case0 = 0;
  Tick Case1 = 0;
  Tick Case2 = 0;
  Tick Case3 = 0;
  Tick Case4 = 0;
  Tick Case5 = 0;
  float *inputPtr, *output;
  int *fetched_inst_num = new int[Total_Trace];
  int *fetched = new int[Total_Trace];
  int *ROB_flag = new int[Total_Trace];
  float *trace_all[Total_Trace];
  Tick *aux_trace_all[Total_Trace];
  //printf("variable init\n");
#pragma omp parallel for
for(int i = 0; i < Total_Trace; i++) {
    int offset = i * Batch_size;
    trace_all[i]= trace + offset * TRACE_DIM;
    aux_trace_all[i]= aux_trace + offset * AUX_TRACE_DIM;
     }
 // printf("Allocated. \n");
  //return 0;
  float *factor_d, *default_val_d, *mean_d;
  float *train_data;
  Tick *curTick,*lastFetchTick;
  //train_data= (float*) malloc(Total_Trace*TD_SIZE*sizeof(float));
  H_ERR(hipMalloc((void **)&curTick, sizeof(Tick)*Total_Trace));
  H_ERR(hipMalloc((void **)&lastFetchTick, sizeof(Tick)*Total_Trace));
  H_ERR(hipMalloc((void **)&output, sizeof(float)*Total_Trace*2));
  hipMemset(curTick, 0, Total_Trace);
  hipMemset(lastFetchTick, 0, Total_Trace);
  hipHostAlloc((void**)&train_data, Total_Trace*INST_SIZE*sizeof(float),
		          hipHostMallocDefault);

  ROB_d rob=ROB_d(Total_Trace);
  H_ERR(hipMalloc((void **)&inputPtr, sizeof(float)*ML_SIZE*Total_Trace));
  //printf("Total mem: %d\n",ML_SIZE*Total_Trace);
  H_ERR(hipMalloc((void **)&rob_d, sizeof(ROB_d)));
  //H_ERR(hipMalloc((void **)&insts, sizeof(float)*Total_Trace*ROB_SIZE*INST_SIZE));
  H_ERR(hipMalloc((void **)&factor_d, sizeof(float)*(TD_SIZE)));
  H_ERR(hipMalloc((void **)&mean_d, sizeof(float)*(TD_SIZE)));
  H_ERR(hipMalloc((void **)&default_val_d, sizeof(float)*(TD_SIZE)));
  H_ERR(hipMalloc((void **)&output, sizeof(float)*(TD_SIZE)*2));
  H_ERR(hipMemcpy(rob_d, &rob, sizeof(ROB_d), hipMemcpyHostToDevice));
  H_ERR(hipMemcpy(factor_d, &factor, sizeof(float)*TD_SIZE, hipMemcpyHostToDevice));
  H_ERR(hipMemcpy(default_val_d, &default_val, sizeof(float)*TD_SIZE, hipMemcpyHostToDevice));
  H_ERR(hipMemcpy(mean_d, &mean, sizeof(float)*TD_SIZE, hipMemcpyHostToDevice));
  //H_ERR(hipMemcpy(mean_d, &mean, sizeof(float)*TD_SIZE*2, hipMemcpyHostToDevice));
  buffers[0]= inputPtr;
  buffers[1]= output; 
  struct timeval start, end, total_start, total_end;
  Inst Inst_;
  float* train_data_d, *insts;
  H_ERR(hipMalloc((void **)&train_data_d, sizeof(float)*Total_Trace*INST_SIZE));
  H_ERR(hipMalloc((void **)&insts, sizeof(float)*Total_Trace*ROBSIZE*INST_SIZE));
  int iteration=0;
  gettimeofday(&total_start, NULL);
  double start_= wtime();
  while(iteration<Batch_size){
    cout<< "Iteration: "<<iteration<<endl;
    double st= wtime(); 
    #pragma omp parallel for
   for(int i=0; i< Total_Trace; i++){ 
    	Inst newInst(train_data);   
          if(!newInst.read_sim_mem(trace_all[i],aux_trace_all[i],train_data,i)){
		cout<<"Inside 1st\n";
       	  }
	  trace_all[i]+=TRACE_DIM;
	  aux_trace_all[i]+=AUX_TRACE_DIM;
      }	 
      display(train_data,INST_SIZE,1);
      double check1= wtime();
      H_ERR(hipMemcpy(train_data_d, train_data, sizeof(float)*Total_Trace*INST_SIZE, hipMemcpyHostToDevice));
      
      double check2= wtime(); 
      preprocess<<<1,32>>>(rob_d, insts,factor_d, mean_d, default_val_d,inputPtr,train_data_d, curTick, lastFetchTick, Total_Trace);
      H_ERR(hipDeviceSynchronize());		
      	double check3= wtime();
	//context->enqueue(Total_Trace, buffers.data(), 0, nullptr); 
        context->enqueueV2(buffers.data(),0,nullptr); 
	hipStreamSynchronize(0);
	update<<<1,32>>>(rob_d, output, insts, factor_d, mean_d, curTick, lastFetchTick, Total_Trace);
        H_ERR(hipDeviceSynchronize());
	iteration++;     
   }
   double end_= wtime();
   for (void* buf : buffers){
	hipFree(buf);
   }
  gettimeofday(&total_end, NULL);
  result<<<1,1>>>(curTick, Total_Trace);
  H_ERR(hipDeviceSynchronize());
  double total_time = total_end.tv_sec - total_start.tv_sec + (total_end.tv_usec - total_start.tv_usec) / 1000000.0;
  cout << "Total time: "<<(end_-start_)<<endl;
#ifdef RUN_TRUTH
  cout << "Truth" << "\n";
#endif
  cout << Instructions << " instructions finish by " << (curTick - 1) << "\n";
  cout << "Time: " << total_time << "\n";
  cout << "MIPS: " << Instructions / total_time / 1000000.0 << "\n";
  cout << "USPI: " << total_time * 1000000.0 / Instructions << "\n";
  cout << "Measured Time: " << measured_time / Instructions << "\n";
  cout << "Cases: " << Case0 << " " << Case1 << " " << Case2 << " " << Case3 << " " << Case4 << " " << Case5 << "\n";
  cout << "Trace: " << argv[1] << "\n";
#ifdef CLASSIFY
  cout << "Model: " << argv[3] << " " << argv[4] << "\n";
#else
  //cout << "Lat Model: " << argv[3] << "\n";
#endif
  return 0;
}

