#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstring>
#include <cassert>
#include <cmath>
#include <sys/time.h>
#include <omp.h>
//#include "init.cuh"
//#include <torch/script.h> // One-stop header.
#include "wtime.h"
using namespace std;
#define NO_MEAN
#define GPU
typedef long unsigned Tick;
typedef long unsigned Addr;
#define TD_SIZE 51
#define ROBSIZE 400
#include "herror.h"
#define MAXSRCREGNUM 8
#define MAXDSTREGNUM 6
#define TD_SIZE 51
#define INST_SIZE 62
#define CONTEXTSIZE 111
#define ROBSIZE 400
#define TICK_STEP 500.0
#define FETCH_BANDWIDTH 3
#define RETIRE_BANDWIDTH 4
#define ML_SIZE (TD_SIZE * CONTEXTSIZE)
#define MIN_COMP_LAT 6
#define WARP
#define ILINEC_BIT 33
#define IPAGEC_BIT 38
#define DADDRC_BIT 42
#define DLINEC_BIT 43
#define DPAGEC_BIT 47
#define PC 51
#define ISADDR 52
#define ADDR 53
#define ADDREND 54
#define IWALK0 55
#define IWALK1 56
#define IWALK2 57
#define DWALK0 58
#define DWALK1 59
#define DWALK2 60
#define COMPLETETICK 61
#define WARPSIZE 32
#define TRACE_DIM 39
#define AUX_TRACE_DIM 10
//#define Total_Trace 1024

Tick Num = 0;

float factor[TD_SIZE];
float mean[TD_SIZE];
float default_val[TD_SIZE];

struct params{
   bool is_empty;
   bool is_full;
   int saturated;
   Tick fetched;
   Tick completeTick;
   bool eof;
   int ROB_flag;
};

class Inst {
	public:
	float *train_data;
  Tick inTick;
  Tick completeTick;
  Tick tickNum;
  Tick trueFetchTick;
  Tick trueCompleteTick;
  int trueFetchClass;
  int trueCompleteClass;
  Addr pc;
  int isAddr;
  Addr addr;
  Addr addrEnd;
  Addr iwalkAddr[3];
  Addr dwalkAddr[3];
  int offset;
  //H_ERR(hipMalloc((void **)&train_data_d, sizeof(int)*TD_SIZE));
  // Read simulation data.
  Inst(){} 
  Inst(float *pointer){
	train_data= pointer;
   }
  bool read_sim_data(ifstream &trace, ifstream &aux_trace, float *train_d, int index) { 
    train_data= train_d;
    trace >> trueFetchClass >> trueFetchTick;
    trace >> trueCompleteClass >> trueCompleteTick;
    aux_trace >> pc;
    if (trace.eof()) {
      assert(aux_trace.eof());
      return false;
    }
    offset= INST_SIZE * index;
    //cout<< "Offset: "<< offset <<"   Memory: "<<train_data;
    assert(trueCompleteTick >= MIN_COMP_LAT);
    for (int i = 4; i < TD_SIZE; i++) {
      trace >> train_data[i+offset];
      train_data[i+offset] /= factor[i];
    }
    train_data[0 + offset] = train_data[1 + offset] = 0.0;
    train_data[2 + offset] = train_data[3 + offset] = 0.0;
    aux_trace >> isAddr >> addr >> addrEnd;
    for (int i = 0; i < 3; i++)
      aux_trace >> iwalkAddr[i];
    for (int i = 0; i < 3; i++)
      aux_trace >> dwalkAddr[i];
    train_data[PC]=pc;
    train_data[ISADDR]=isAddr;
    train_data[ADDR]=addr;
    train_data[ADDREND]=addrEnd;
    train_data[IWALK0]=iwalkAddr[0];
    train_data[IWALK1]=iwalkAddr[1];
    train_data[IWALK2]=iwalkAddr[2];
    train_data[DWALK0]=dwalkAddr[0];
    train_data[DWALK1]=dwalkAddr[1];
    train_data[DWALK2]=dwalkAddr[2];
    assert(!trace.eof() && !aux_trace.eof());
    //cout << "in: ";
    //for (int i = 0; i < TD_SIZE; i++)
    //  cout << train_data[i] << " ";
    //cout << "Read complete\n";
    //H_ERR(hipMemcpy(train_data_d, train_data, sizeof(float)*TD_SIZE, hipMemcpyHostToDevice));
    return true;
  }

 bool read_sim_mem(float *trace, uint64_t *aux_trace, float *train_d, int index) {
    train_data= train_d;
    //trace >> trueFetchClass >> trueFetchTick;
    //trace >> trueCompleteClass >> trueCompleteTick;
    trueFetchClass= trace[0];
    trueFetchTick= trace[1];
    trueCompleteClass= trace[2];
    trueCompleteTick= trace[3];
    pc= aux_trace[0];
    
    
    offset= INST_SIZE * index;
    //cout<< "Offset: "<< offset <<"   Memory: "<<train_data;
    //assert(trueCompleteTick >= MIN_COMP_LAT);
    for (int i = 4; i < TD_SIZE; i++) {
      train_data[i+offset]= trace[i]/factor[i];
      //train_data[i+offset] /= factor[i];
    }
    train_data[0 + offset] = train_data[1 + offset] = 0.0;
    train_data[2 + offset] = train_data[3 + offset] = 0.0;
    //aux_trace >> isAddr >> addr >> addrEnd;
    isAddr= aux_trace[1];
    addr= aux_trace[2];
    addrEnd= aux_trace[3];
    for (int i = 0; i < 3; i++)
      iwalkAddr[i]=aux_trace[3+i];
    for (int i = 0; i < 3; i++)
      dwalkAddr[i]=aux_trace[6+i];
    train_data[PC]=pc;
    train_data[ISADDR]=isAddr;
    train_data[ADDR]=addr;
    train_data[ADDREND]=addrEnd;
    train_data[IWALK0]=iwalkAddr[0];
    train_data[IWALK1]=iwalkAddr[1];
    train_data[IWALK2]=iwalkAddr[2];
    train_data[DWALK0]=dwalkAddr[0];
    train_data[DWALK1]=dwalkAddr[1];
    train_data[DWALK2]=dwalkAddr[2];
    //assert(!trace.eof() && !aux_trace.eof());
    return true;
 }	
};

class ROB{
public:
    //Inst insts[ROBSIZE+1];
    float *insts;
    int head= 0;
    int tail= 0;
    bool saturated= false; 
    void init(){
        //insts.init();
        //H_ERR(hipMalloc((void **)&train_data, sizeof(int)*(ROBSIZE +1)));
    }
    ~ROB(){};
    ROB(){
        H_ERR(hipMalloc((void **)&insts, sizeof(float)*(ROBSIZE*INST_SIZE)));
    };
    __host__ __device__ int inc(int input) {
        if (input == ROBSIZE)
          return 0;
        else
          return input + 1;
    }

    __host__ __device__ int dec(int input) {
        if (input == 0)
          return ROBSIZE;
        else
          return input - 1;
    }
    __host__ __device__ bool is_empty() { return head == tail; }
    __host__ __device__ bool is_full() { return head == inc(tail); }

__host__ __device__
     void add() {
    assert(!is_full());
    int old_tail = tail;
    tail = inc(tail);
    //printf("index updated.\n");
    //return &insts[old_tail];
  }
    /*
    __device__
    Inst *getHead() {
        return &insts[head];
      }
      */

__device__ void
	retire(){
		assert(!is_empty());
		head= inc(head);
	}

 __device__
 int retire_until(Tick tick, float *insts) {
	//printf("Head: %d\n",head);
	 int completeTick;
	int retired = 0;
	while (!is_empty() && insts[COMPLETETICK] <= tick) {
		retire();
		retired++;
	}
	return retired;
 }

/*
	  __device__
    void update_fetch_cycle(Tick tick, Tick curTick, float *factor) {
        int TID= (blockIdx.x * blockDim.x) + threadIdx.x;
	//int warpID= TID / WARPSIZE;
	int  warpTID= TID/ WARPSIZE;

    	assert(!is_empty());
        int start = dec(dec(tail));
        int end= dec(head);
        //for (int i = dec(dec(tail)); i != dec(head); i = dec(i)) {
        //printf("start: %d, end: %d\n",start,end);
        int i= (start - warpTID); 
	
	//for (int i = dec(dec(tail)); i != dec(head); i = dec(i)) {
          while(i>end){
		if (insts[i].completeTick <= curTick)
            		continue;
          insts[i].train_data[0] += tick / factor[0];
          if (insts[i].train_data[0] >= 9 / factor[0])
            insts[i].train_data[0] = 9 / factor[0];
          insts[i].train_data[1] += tick / factor[1];
          assert(insts[i].train_data[0] >= 0.0);
          assert(insts[i].train_data[1] >= 0.0);
        }
      }

*/      
__device__ 
	  int make_input_data(float *context, float *insts, Tick tick, float *factor, float *default_val) {
 	//if(){printf("Here. Head: %d, Tail: %d\n",head,tail);}

 	int TID= (blockIdx.x * blockDim.x) + threadIdx.x;
	int warpID= TID / WARPSIZE;
	int  warpTID= TID % WARPSIZE;
	int offset;
#ifdef WARP
	offset= warpID;
#else
	offset= blockIdx.x;
#endif
 	int curr= dec(tail);
	int start_context= dec(dec(tail));
	int end_context= dec(head);
	//insts= insts + offset + INST_SIZE;
	//if(warpTID==0){printf("Here. Head: %d, Tail: %d\n",head,tail);}
	assert(!is_empty());
        saturated = false;
	__shared__ int num[4];
        Addr pc = insts[curr * INST_SIZE + PC];
        int isAddr= insts[curr * INST_SIZE + ISADDR];
        Addr addr = insts[curr * INST_SIZE + ADDR];
        Addr addrEnd = insts[curr * INST_SIZE + ADDREND];
        Addr iwalkAddr[3], dwalkAddr[3];
        int i= warpTID;
	//if (warpTID==0){
	while(i<3){
	//for (int i = 0; i < 3; i++) {
          iwalkAddr[i] = insts[curr*INST_SIZE + IWALK0 + i];
          dwalkAddr[i] = insts[curr*INST_SIZE + DWALK0 + i];
        i++;
	}
	__syncwarp();
	
	//int start = dec(dec(tail)); int end= dec(head);
        //int num= end-start;
        if(warpTID==0){printf("Here. Head: %d, Tail: %d,current:%d, start: %d, end: %d, curr: %d \n",head,tail,curr,start_context,end_context,dec(tail));}	
	i= start_context - warpTID;
	while(i > end_context){  
	  printf("ThreadID: %d, inst id: %d\n",warpTID, i);
	  if (insts[i*INST_SIZE+COMPLETETICK] <= tick)
            continue;
          if (num[warpID] >= CONTEXTSIZE) {
            saturated = true;
            return 0;
          }
          // Update context instruction bits.
          insts[i*INST_SIZE+ ILINEC_BIT] = insts[i*INST_SIZE+PC] == pc ? 1.0 / factor[ILINEC_BIT] : 0.0;
          int conflict = 0;
          for (int j = 0; j < 3; j++) {
            if (insts[i*INST_SIZE+j] != 0 && insts[i*INST_SIZE+j] == iwalkAddr[j])
              conflict++;}
          //insts[i].train_data[IPAGEC_BIT] = (float)conflict / factor[IPAGEC_BIT];
          //insts[i].train_data[DADDRC_BIT] = (isAddr && insts[i].train_data[ISADDR] && addrEnd >= insts[i].train_data[ADDR] && addr <= insts[i].train_data[ADDREND]) ? 1.0 / factor[DADDRC_BIT] : 0.0;
          //insts[i].train_data[DLINEC_BIT] = (isAddr && insts[i].train_data[ISADDR] && (addr & ~0x3f) == (insts[i].train_data[ADDR] & ~0x3f)) ? 1.0 / factor[DLINEC_BIT] : 0.0;
          conflict = 0;
          if (isAddr && insts[i*INST_SIZE+ISADDR])
            for (int j = 0; j < 3; j++) {
              if (insts[i*INST_SIZE+j] != 0 && insts[i*INST_SIZE+j] == dwalkAddr[j])
                conflict++;}
          insts[i*INST_SIZE+DPAGEC_BIT] = (float)conflict / factor[DPAGEC_BIT];
          //std::copy(insts[i].train_data, insts[i].train_data + TD_SIZE, context + num * TD_SIZE);
          //num++;
	  atomicAdd(&num[warpID],1);
        i-=WARPSIZE;
	}
	__syncwarp();
       //if(warpTID==0){printf("Here. Head: %d, Tail: %d, start: %d, end: %d, curr: %d \n",head,tail,start,end,dec(tail));}	
	/* Data copy: current instruction and ROB instructions*/
	/*
	int j= warpTID;
	while(j>=end_context)
	{
		if(warpTID==0){ printf("Working on context: %d\n",j);}
		i= warpTID;	 
		while(i<TD_SIZE){
			context[i]= insts[j*INST_SIZE+i];
			i+=WARPSIZE;
		}
		j-=1;
	}
	*/
        i= warpTID;
        while (i<TD_SIZE){
                //for (int i = num; i < CONTEXTSIZE; i++) { //printf("thread: %d, i: %d\n",warpTID,i);
                //if(warpTID==0){printf("");}
                int j= curr;
                while(j!= end_context){
                        context[i+j*TD_SIZE]= insts[j*INST_SIZE+i];
                        //printf("Context: %d, index: %d,pos: %d, thread: %d, write: %.2f\n", j,i,i+j*TD_SIZE,warpTID, default_val[i]);
                        j=dec(j);}
        i+=WARPSIZE;}
	__syncwarp();

	//printf("Adding default values.\n");
	i= warpTID;
	while (i<TD_SIZE){
        	//for (int i = num; i < CONTEXTSIZE; i++) { //printf("thread: %d, i: %d\n",warpTID,i);
		//if(warpTID==0){printf("");}
		int j= 1;
		while(j< CONTEXTSIZE){
			context[i+j*TD_SIZE]= default_val[i];
			//printf("Context: %d, index: %d,pos: %d, thread: %d, write: %.2f\n", j,i,i+j*TD_SIZE,warpTID, default_val[i]);
			j++;}
	i+=WARPSIZE;}
	__syncwarp();
	return 0;
      }
};


class ROB_d {
   public:
	ROB *rob;
       ROB_d(int Total_Trace){
       		//ROB rob[Total_Trace]; 		
		H_ERR(hipMalloc((void **)&rob, sizeof(ROB)*(Total_Trace)));

       }	
};

__device__ void
dis(float *data, int size, int rows)
{
	for(int i=0;i<rows;i++)
	{
		for(int j=0; j<size;j++){
		printf("%.1f  ",data[i*size+j]);
		}
		printf("\n");
	}
}

__global__ void
preprocess(ROB_d *rob_d, float *insts,  float *factor, float *mean, float *default_val, float *inputPtr, float *train_data, params *param, int Total_Trace )
{
    
    int fetched=0, curTick=0, lastFetchTick=0;
    int TID=(blockIdx.x * blockDim.x) + threadIdx.x ;
    int warpID= TID/WARPSIZE;
    int warpTID = TID%WARPSIZE;
    int TotalWarp = (gridDim.x * blockDim.x) / WARPSIZE;
    int index,Total;
    ROB *rob;
    float *rob_pointer;
    float *input_ptr; 
#ifdef WARP	
    index= warpID;
    Total= TotalWarp;
#else
    index= blockIdx.x;
    Total= gridDim.x;
#endif
     while(index<Total_Trace){
     	rob = &rob_d->rob[index];
    //if(warpTID==0) { printf("Read: Warp: %d, assigned: %d, next: %d\n",warpID, index, index + Total);}
    //push new instruction to respective ROB but not latency
    int tail= rob->dec(tail);
    //if(warpTID==0) { printf("Read: Warp: %d, assigned: %d, next: %d\n",warpID, index, index + Total);}
    //int tail= rob->dec(tail);
    rob_pointer= insts + ROBSIZE * INST_SIZE * index;	
     float *input_Ptr = inputPtr + ML_SIZE * index;
    int i= warpTID+4; 
    while(i<INST_SIZE)
    {
	    rob_pointer[i]= train_data[i + warpID * INST_SIZE];
	    //printf("t: %d, i: %d, offset: %d\n",TID,i,train_offset);	
	    i+=WARPSIZE;		
    }
    __syncwarp();
    //rob = &rob_d->rob[TID];
    //if(warpTID==0) { printf("Inpt: %d\n",warpID);} 
    if(warpTID==0){
       	int retired = rob->retire_until(curTick, insts); 
    	//printf("Retired. \n");
	fetched++;	
    	rob->add();
	//printf("Update: ROB: %d, thread: %d, head:%d, tail: %d, newIndex: %d\n", index, threadIdx.x, rob->head, rob->tail, (index + gridDim.x * blockDim.x));
    	if (curTick != lastFetchTick) {
        	//rob->update_fetch_cycle(curTick - lastFetchTick, curTick, factor);
   	}
    }
    __syncwarp();
    //if(TID==0){printf("update completed\n"); }
    //rob = &rob_d->rob[index]; 
    //while(index<Total_Trace){
	//if(warpTID==0) { printf("Make input: Warp: %d, assigned: %d,offset: %d, next: %d\n",warpID, index,ML_SIZE*index, index + Total);}
    	rob->make_input_data(input_Ptr, rob_pointer, curTick, factor, default_val);        
    	index+= Total;  
	if(warpTID==0){
		printf("Input_Ptr\n");
		dis(input_Ptr,TD_SIZE,3 );
	}	
    }
}

__global__ void
update( ROB_d *rob_d, float* output, float* inputPtr, float* factor, float* mean ){

	      int TID=(blockIdx.x * blockDim.x) + threadIdx.x ;
      	      int offset = TID *2;
	      float fetch_lat = output[offset+0] * factor[1] + mean[1];
	      float finish_lat = output[offset+1] * factor[3] + mean[3];
	      int int_fetch_lat = round(fetch_lat);
	      int int_finish_lat = round(finish_lat);
	      if (int_fetch_lat < 0)
		int_fetch_lat = 0;
	      if (int_finish_lat < MIN_COMP_LAT)
		int_finish_lat = MIN_COMP_LAT;

            inputPtr = inputPtr + ML_SIZE * TID; 
	   inputPtr[0] = (-int_fetch_lat - mean[0]) / factor[0];
 	   inputPtr[1] = (-int_fetch_lat - mean[1]) / factor[1];
	   inputPtr[2] = (int_finish_lat - MIN_COMP_LAT - mean[2]) / factor[2];
	   if (inputPtr[2] >= 9 / factor[2])
	   	inputPtr[2] = 9 / factor[2];
	   inputPtr[3] = (int_finish_lat - mean[3]) / factor[3];
	//newInst->tickNum = int_finish_lat;
	//newInst->completeTick = curTick[0] + int_finish_lat + int_fetch_lat;
	   //int lastFetchTick = curTick;
	   /*
	   if (total_num && fetched_inst_num[0] == total_num) {
		eof[0] = true;
		break;
		}
	if (int_fetch_lat) {
		nextFetchTick = curTick + int_fetch_lat;
		break;
		}


	ROB *rob = &rob_d->rob[TID];
	if (rob->is_empty())
	{

	
	}
	*/
}



void display(float *data, int size, int rows)
{
	for(int i=0;i<rows;i++){
		for(int j=0;j<size;j++){
			printf("%.2f\t",data[i*size+j]);
		}
		printf("\n");
	}
}

void display(unsigned long *data, int size, int rows)
{
	        for(int i=0;i<rows;i++){
			                for(int j=0;j<size;j++){
						                        printf("%.ld\t",data[i*size+j]);
									                }
					                printf("\n");
							        }
}



float *read_numbers(char *fname, int sz) {
  float *ret = new float[sz];
  ifstream in(fname);
  //printf("Trying to read from %s\n", fname);
  for(int i=0;i<sz;i++)
    in >> ret[i];
  return ret;
}

int read_trace_mem(char trace_file[], char aux_trace_file[], float *trace, unsigned long *aux_trace, int instructions)
{
  FILE *trace_f=fopen(trace_file,"rb");
  if(!trace_f){
	printf("Unable to read trace binary.");
	return 1;
	}
    int r=fread(trace,sizeof(float),TRACE_DIM*instructions,trace_f);
    printf("read :%d values for trace.\n",r);
    //display(trace,TRACE_DIM,2);

  FILE *aux_trace_f=fopen(aux_trace_file,"rb");
  if(!aux_trace_f){
        printf("Unable to aux_trace binary.");
        return 1;
        }
    int k=fread(aux_trace,sizeof(unsigned long),AUX_TRACE_DIM*instructions,aux_trace_f);  
    printf("read :%d values for aux_trace.\n",k);
    display(aux_trace,AUX_TRACE_DIM,2);
}

int main(int argc, char *argv[]) {
printf("args count: %d\n",argc);
#ifdef CLASSIFY
  if (argc != 8) {
    cerr << "Usage: ./simulator_q <trace> <aux trace> <lat module> <class module> <variances> <# inst> <Total trace>" << endl;
    return 0;
  }
#else
  if (argc != 7) {
    cerr << "Usage: ./simulator_q <trace> <aux trace> <lat module> <variances> <Total trace> <#Insts>" << endl;
#endif
    return 0;
  } 
  int arg_idx=4;
  float *varPtr = read_numbers(argv[arg_idx++], TD_SIZE);
  for (int i = 0; i < TD_SIZE; i++) {
#ifdef NO_MEAN
    mean[i] = -0.0;
#endif
    factor[i] = sqrtf(varPtr[i]);
    default_val[i] = -mean[i] / factor[i]; 
    }
  int Total_Trace= atoi(argv[arg_idx++]);
  int Instructions= atoi(argv[arg_idx++]);  
  
  float *trace;
  unsigned long *aux_trace;
  trace=(float*) malloc(TRACE_DIM*Instructions*sizeof(float));
  aux_trace=(unsigned long*) malloc(AUX_TRACE_DIM*Instructions*sizeof(unsigned long));
  read_trace_mem(argv[1],argv[2],trace,aux_trace,Instructions); 
  int Batch_size = Instructions / Total_Trace;
  int stop_flag, inst_num;
  //cout << "Batch size:  "<<Batch_size<<endl;
  //cout<<"Parameters read..\n";
 
   omp_set_num_threads(96);
   double measured_time = 0.0;
 
  ROB_d  *rob_d;
    Tick Case0 = 0;
  Tick Case1 = 0;
  Tick Case2 = 0;
  Tick Case3 = 0;
  Tick Case4 = 0;
  Tick Case5 = 0;
  float *inputPtr;
  //ifstream *trace = new ifstream[Total_Trace];
  //ifstream *aux_trace = new ifstream[Total_Trace];
  Tick *curTick = new Tick[Total_Trace];
  Tick *nextFetchTick = new Tick[Total_Trace];
  Tick *lastFetchTick = new Tick[Total_Trace];
  int *index = new int[Total_Trace];
  int *inst_num_all = new int[Total_Trace];
  int *fetched_inst_num = new int[Total_Trace];
  int *fetched = new int[Total_Trace];
  int *ROB_flag = new int[Total_Trace];
  int *int_fetch_latency = new int[Total_Trace];
  int *int_finish_latency = new int[Total_Trace];
  bool *eof = new bool[Total_Trace];
  int total_num= 10000;
  float *trace_all[Total_Trace];
  unsigned long *aux_trace_all[Total_Trace];
  //printf("variable init\n");
  
#pragma omp parallel for
for(int i = 0; i < Total_Trace; i++) {
    curTick[i] = 0;
    nextFetchTick[i] = 0;
    lastFetchTick[i] = 0;
    inst_num_all[i] = 0;
    fetched_inst_num[i] = 0;
    fetched[i] = 0;
    eof[i] = false;
    int offset = i * Batch_size;
    trace_all[i]= trace + offset * TRACE_DIM;
    aux_trace_all[i]= aux_trace + offset * AUX_TRACE_DIM;
    //std::string line, line1;
    int number_of_lines = 0;
     }
 // printf("Allocated. \n");
  //return 0;
  float *factor_d, *default_val_d, *mean_d;
  float* train_data;
  //train_data= (float*) malloc(Total_Trace*TD_SIZE*sizeof(float));
  hipHostAlloc((void**)&train_data, Total_Trace*INST_SIZE*sizeof(float),
		          hipHostMallocDefault);
  //printf("before rob\n");
  //return 0;
  ROB_d rob=ROB_d(Total_Trace);
  //printf("rob allocated\n");
  //return 0;
  //cout<<"Rob tail: "<<rob.tail<<"\n"; 
  H_ERR(hipMalloc((void **)&inputPtr, sizeof(float)*ML_SIZE*Total_Trace));
  //printf("Total mem: %d\n",ML_SIZE*Total_Trace);
  H_ERR(hipMalloc((void **)&rob_d, sizeof(ROB_d)));
  //H_ERR(hipMalloc((void **)&insts, sizeof(float)*Total_Trace*ROB_SIZE*INST_SIZE));
  H_ERR(hipMalloc((void **)&factor_d, sizeof(float)*(TD_SIZE)));
  H_ERR(hipMalloc((void **)&mean_d, sizeof(float)*(TD_SIZE)));
  H_ERR(hipMalloc((void **)&default_val_d, sizeof(float)*(TD_SIZE)));
  H_ERR(hipMemcpy(rob_d, &rob, sizeof(ROB_d), hipMemcpyHostToDevice));
  H_ERR(hipMemcpy(factor_d, &factor, sizeof(float)*TD_SIZE, hipMemcpyHostToDevice));
  H_ERR(hipMemcpy(default_val_d, &default_val, sizeof(float)*TD_SIZE, hipMemcpyHostToDevice));
  H_ERR(hipMemcpy(mean_d, &mean, sizeof(float)*TD_SIZE, hipMemcpyHostToDevice));
  struct timeval start, end, total_start, total_end;
  //printf("Cuda allocated\n");
  //return 0;
  gettimeofday(&total_start, NULL);
  bool is_empty=true;
  bool is_full=false;
  bool saturated=false;
  Tick retired=0;
  Tick completeTick=0;
  //cout<<"Loop starting....\n";
  Inst Inst_;
  struct params Host, *Device;
  struct Inst *newInst;
  float* train_data_d, *insts;
  H_ERR(hipMalloc((void **)&train_data_d, sizeof(float)*Total_Trace*INST_SIZE));
  H_ERR(hipMalloc((void **)&insts, sizeof(float)*Total_Trace*ROBSIZE*INST_SIZE));
  H_ERR(hipMalloc((void **)&Device, sizeof(params)));
  //printf("InputPtr: %d, Train:%d\n ",Total_Trace*ML_SIZE,Total_Trace*TD_SIZE); 
  //H_ERR(hipMalloc((void **)&Inst_.train_data_d, sizeof(float)*TD_SIZE));
  while(stop_flag!=1){
   double st= wtime(); 
    #pragma omp parallel for
   for(int i=0; i< Total_Trace; i++){
    	// Retire instructions.
    	inst_num += retired;
    	int fetched = 0;
    	int int_fetch_lat;
    	//int i=0;
    	//cout<<"First loop.i:" <<i<<endl; 
    	Inst newInst(train_data);    
    	//double st=wtime();
    	//trace+=((i%512)*39);
	//if (!newInst.read_sim_data(trace[i], aux_trace[i], train_data, i)) {
          if(!newInst.read_sim_mem(trace_all[i],aux_trace_all[i],train_data,i)){
		eof[i] = true;
		cout<<"Inside 1st\n";
        	//rob->tail = rob->dec(rob->tail);
      	}
	  trace_all[i]+=TRACE_DIM;
	  aux_trace_all[i]+=AUX_TRACE_DIM;

      }	 
      //display(train_data,INST_SIZE,2);
      double check1= wtime();
      H_ERR(hipMemcpy(train_data_d, train_data, sizeof(float)*Total_Trace*INST_SIZE, hipMemcpyHostToDevice));
      double check2= wtime();
      /*
      for(int i=0; i<TD_SIZE;i++)
      {
	      printf("%.3f\t",Inst_.train_data[i]);
	      if(i%10==0)
		      printf("\n");
      }
      printf("calling gpu function\n");
	*/
        int block= Total_Trace/2;

        preprocess<<<1,32>>>(rob_d, insts,factor_d, mean_d, default_val_d,inputPtr,train_data_d, Device, Total_Trace);
	H_ERR(hipDeviceSynchronize());		
      	double en= wtime(); 
	printf("%d, %.6f, %.6f, %.6f, %.6f\n", Total_Trace,(check1-st),(check2-check1),(en-check2),(en-st));
        return 0;	
	H_ERR(hipMemcpy(&Host, Device, sizeof(params), hipMemcpyDeviceToHost));
	//cout<<"Here\n";
	is_empty= Host.is_empty;
	is_full= Host.is_full;
	saturated= Host.saturated;
	//cout<<"Done\n";
	//return 0;
	 float output[]={1.5,3.20,0,0,0,0};
      	measured_time += (end.tv_sec - start.tv_sec) * 1000000.0 + end.tv_usec - start.tv_usec;
      //cout << 1000000 * (end.tv_sec - start.tv_sec) + end.tv_usec - start.tv_usec << "\n";
  }
  gettimeofday(&total_end, NULL);
  double total_time = total_end.tv_sec - total_start.tv_sec + (total_end.tv_usec - total_start.tv_usec) / 1000000.0;

  //trace[0].close();
  //aux_trace[0].close();
#ifdef RUN_TRUTH
  cout << "Truth" << "\n";
#endif
  cout << inst_num << " instructions finish by " << (curTick - 1) << "\n";
  cout << "Time: " << total_time << "\n";
  cout << "MIPS: " << inst_num / total_time / 1000000.0 << "\n";
  cout << "USPI: " << total_time * 1000000.0 / inst_num << "\n";
  cout << "Measured Time: " << measured_time / inst_num << "\n";
  cout << "Cases: " << Case0 << " " << Case1 << " " << Case2 << " " << Case3 << " " << Case4 << " " << Case5 << "\n";
  cout << "Trace: " << argv[1] << "\n";
#ifdef CLASSIFY
  cout << "Model: " << argv[3] << " " << argv[4] << "\n";
#else
  cout << "Lat Model: " << argv[3] << "\n";
#endif
  return 0;
}

