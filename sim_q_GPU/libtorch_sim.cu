#include "hip/hip_runtime.h"
#include <memory>
#include <vector>
#include <iostream>
#include <fstream>
#include <cstring>
#include <cassert>
#include <cmath>
#include <sys/time.h>
#include <stdio.h>
#include <omp.h>
#include "wtime.h"
#include "herror.h"
//#include "trt.cuh"
#include "sim.cuh"
#include<torch/script.h>
using namespace std;
#define NO_MEAN
#define GPU
#define WARP
//#define DEBUG

//#define Total_Trace 1024

Tick Num = 0;



__global__ void
preprocess(ROB *rob_d, Inst *insts, float *factor, float *mean, float *default_val, float *inputPtr, int *status, int Total_Trace)
{
  int TID = (blockIdx.x * blockDim.x) + threadIdx.x;
  int warpID = TID / WARPSIZE;
  int warpTID = TID % WARPSIZE;
  int TotalWarp = (gridDim.x * blockDim.x) / WARPSIZE;
  int index, Total;
  ROB *rob;
  float *input_Ptr;
#ifdef WARP
  index = warpID;
  Total = TotalWarp;
#else
  index = blockIdx.x;
  Total = gridDim.x;
#endif
  while (index < Total_Trace)
  {
    rob= &rob_d[index];
#ifdef DEBUG
    if(threadIdx.x==0){ printf("Before memcpy: Head: %d,Tail: %d, len: %d,\n", rob->head,rob->tail, rob->len);}
#endif
    Tick curTick = rob->curTick;
    Tick lastFetchTick = rob->lastFetchTick;
    input_Ptr= inputPtr + ML_SIZE * index;  
    //int old_head= rob->head;

if (warpTID == 0)
    {
      if (status[index]==1)
      {      
              int retired = rob->retire_until(curTick); 
#ifdef DEBUG
              printf("Retire until: %ld, Retired: %d\n",curTick, retired);
#endif
      }
    }
	__syncwarp();
    if(warpTID==0){
	     Inst *newInst = rob->add();
	    //printf("Rob pointer before: %p, new Inst: %p, head: %d\n",rob,newInst,rob->dec(rob->tail));
	    memcpy(newInst, &insts[index], sizeof(Inst));
    	    //inst_copy(&rob->insts[rob->tail],&insts[index]);  
#ifdef DEBUG
	    printf("Rob pointer after: %p, new Inst: %p, head: %d\n",rob,newInst,rob->dec(rob->tail));
#endif
    }
    __syncwarp();
    //printf("Curtick: %ld, lastFetchTick: %ld\n", curTick, lastFetchTick);
    if (curTick != lastFetchTick)
    {
      rob->update_fetch_cycle(curTick - lastFetchTick, factor);
    } 
    __syncwarp();
    rob->make_input_data(input_Ptr,curTick, factor, default_val);
#ifdef DEBUG
    if (warpTID == 0)
    {
      printf("Input_Ptr\n");
      dis(input_Ptr, TD_SIZE, 6);
    }
#endif
    __syncwarp();
    index += Total;
  }
}


void display(float *data, int size, int rows)
{
  for (int i = 0; i < rows; i++)
  {
    for (int j = 0; j < size; j++)
    {
      printf("%.2f\t", data[i * size + j]);
    }
    printf("\n");
  }
}

void display(unsigned long *data, int size, int rows)
{
  for (int i = 0; i < rows; i++)
  {
    for (int j = 0; j < size; j++)
    {
      printf("%.f\t", (float)data[i * size + j]);
    }
    printf("\n");
  }
}

float *read_numbers(char *fname, int sz)
{
  float *ret = new float[sz];
  ifstream in(fname);
  //printf("Trying to read from %s\n", fname);
  for (int i = 0; i < sz; i++)
    in >> ret[i];
  return ret;
}

/*
int read_trace_mem(char trace_file[], char aux_trace_file[], float *trace, Tick *aux_trace, int instructions)
{
  FILE *trace_f = fopen(trace_file, "rb");
  if (!trace_f)
  {
    printf("Unable to read trace binary.");
    return 1;
  }
  Tick tot= TRACE_DIM * instructions;
  Tick r = fread(trace, sizeof(float), TRACE_DIM * instructions, trace_f);
  printf("tot: %lu, Toread: %lu, read :%lu values for trace.\n",tot,TRACE_DIM * instructions, r);
  //display(trace,TRACE_DIM,2);

  FILE *aux_trace_f = fopen(aux_trace_file, "rb");
  if (!aux_trace_f)
  {
    printf("Unable to aux_trace binary.");
    return 1;
  }
  int k = fread(aux_trace, sizeof(Tick), AUX_TRACE_DIM * instructions, aux_trace_f);
  printf("read :%d values for aux_trace.\n", k);
  //display(aux_trace,AUX_TRACE_DIM,2);
  return true;
}
*/


int main(int argc, char *argv[])
{
  printf("args count: %d\n", argc);
#ifdef CLASSIFY
  if (argc != 8)
  {
    cerr << "Usage: ./simulator_q <trace> <aux trace> <lat module> <class module> <variances> <# inst> <Total trace>" << endl;
    return 0;
  }
#else
  if (argc != 7)
  {
    cerr << "Usage: ./simulator_q <trace> <aux trace> <lat module> <variances> <Total trace> <#Insts>" << endl;
#endif
  return 0;
}
int arg_idx = 4;
float *varPtr = read_numbers(argv[arg_idx++], TD_SIZE);
for (int i = 0; i < TD_SIZE; i++)
{
#ifdef NO_MEAN
  mean[i] = -0.0;
#endif
  factor[i] = sqrtf(varPtr[i]);
  default_val[i] = -mean[i] / factor[i];
  //cout<<default_val[i]<<" ";
}

torch::jit::script::Module lat_module;
  try {
    // Deserialize the ScriptModule from a file using torch::jit::load().
    lat_module = torch::jit::load(argv[3]);
#ifdef GPU
    lat_module.to(torch::kCUDA);
#endif
  }
  catch (const c10::Error& e) {
    cerr << "error loading the model\n";
    return 0;
  }

//cout<<endl;
int Total_Trace = atoi(argv[arg_idx++]);
int Instructions = atoi(argv[arg_idx++]);
cout<< "Total_Trace: "<< Total_Trace << ", Instructions: "<< Instructions << endl;
std::string model_path(argv[3]);
at::Tensor input = torch::ones({Total_Trace, ML_SIZE});
float *inp= input.data_ptr<float>();
//cout<<"Input dims: "<< input_dims << ", output dims: "<<output_dims << endl;

float *inputPtr, *output;
H_ERR(hipMalloc((void **)&inputPtr, sizeof(float) * ML_SIZE * Total_Trace));
H_ERR(hipMalloc((void **)&output, sizeof(float) * Total_Trace * 22));
//cout<< "Input dim: "<< ML_SIZE * Total_Trace << endl;
float *trace;
Tick *aux_trace;
trace = (float *)malloc(TRACE_DIM * Instructions * sizeof(float));
aux_trace = (Tick *)malloc(AUX_TRACE_DIM * Instructions * sizeof(Tick));
read_trace_mem(argv[1], argv[2], trace, aux_trace, Instructions);
int Batch_size = Instructions / Total_Trace;
cout << " Iterations: " << Batch_size << endl;
//cout<<"Parameters read..\n";
omp_set_num_threads(1);
double measured_time = 0.0;
Tick Case0 = 0;
Tick Case1 = 0;
Tick Case2 = 0;
Tick Case3 = 0;
Tick Case4 = 0;
Tick Case5 = 0;
int *fetched_inst_num = new int[Total_Trace];
int *fetched = new int[Total_Trace];
int *ROB_flag = new int[Total_Trace];
float *trace_all[Total_Trace];
Tick *aux_trace_all[Total_Trace];
//printf("variable init\n");
#pragma omp parallel for
for (int i = 0; i < Total_Trace; i++)
{
  int offset = i * Batch_size;
  trace_all[i] = trace + offset * TRACE_DIM;
  aux_trace_all[i] = aux_trace + offset * AUX_TRACE_DIM;
}
// printf("Allocated. \n");
//return 0;
float *factor_d, *default_val_d, *mean_d;
Tick *curTick, *lastFetchTick;
int *status;
H_ERR(hipMalloc((void **)&curTick, sizeof(Tick) * Total_Trace));
H_ERR(hipMalloc((void **)&lastFetchTick, sizeof(Tick) * Total_Trace));
H_ERR(hipMalloc((void **)&status, sizeof(int) * Total_Trace));
hipMemset(curTick, 0, Total_Trace);
hipMemset(lastFetchTick, 0, Total_Trace);
//hipHostAlloc((void **)&train_data, Total_Trace *INST_SIZE * sizeof(float),
  //            hipHostMallocDefault);
struct ROB *rob= new ROB[Total_Trace];
struct Inst *inst= new Inst[Total_Trace];
struct ROB *rob_d;
struct Inst *inst_d;
H_ERR(hipMalloc((void **)&rob_d, sizeof(ROB)*Total_Trace));
H_ERR(hipMalloc((void **)&inst_d, sizeof(Inst)*Total_Trace));
// For factor, mean and default values
H_ERR(hipMalloc((void **)&factor_d, sizeof(float) * (TD_SIZE)));
H_ERR(hipMalloc((void **)&mean_d, sizeof(float) * (TD_SIZE)));
H_ERR(hipMalloc((void **)&default_val_d, sizeof(float) * (TD_SIZE)));
H_ERR(hipMemcpy(factor_d, &factor, sizeof(float) * TD_SIZE, hipMemcpyHostToDevice));
H_ERR(hipMemcpy(default_val_d, &default_val, sizeof(float) * TD_SIZE, hipMemcpyHostToDevice));
H_ERR(hipMemcpy(mean_d, &mean, sizeof(float) * TD_SIZE, hipMemcpyHostToDevice));
struct timeval total_start, total_end;
int iteration = 0;
gettimeofday(&total_start, NULL);
double start_ = wtime();
double red=0,pre=0, tr=0,inf=0,upd=0;
FILE *pFile;
pFile= fopen ("libcustom.bin", "wb");
while (iteration < Batch_size){
  //if((iteration % 50)==0){cout << "Iteration: " << iteration << endl;}
  double st = wtime();
#pragma omp parallel for
  for (int i = 0; i < Total_Trace; i++)
  {
    if (!inst[i].read_sim_mem(trace_all[i], aux_trace_all[i],i))
    {cout << "Error\n";}
    trace_all[i] += TRACE_DIM; aux_trace_all[i] += AUX_TRACE_DIM;
    //printf("Trace: %d, read\n",i);
    } 
  double check1 = wtime();
  red+= (check1-st);
    H_ERR(hipMemcpy(inst_d, inst, sizeof(Inst) * Total_Trace, hipMemcpyHostToDevice));
    double check2 = wtime();
  tr+= (check2-check1);
  preprocess<<<4096, 64>>>(rob_d,inst_d, factor_d, mean_d, default_val_d, inputPtr, status, Total_Trace);
  H_ERR(hipDeviceSynchronize());
  //cout<<"Preprocess done \n"<<endl; 
  double check3= wtime();
    H_ERR(hipMemcpy(inp,inputPtr, sizeof(float) * ML_SIZE*Total_Trace, hipMemcpyDeviceToHost));
  fwrite(inp, sizeof(float), ML_SIZE, pFile);
  //printf("Input:\n");
  //display(inp, 51,4);
  pre+= (check3-check2);
  check3 = wtime();
  //pre+= (check3-check2);
  std::vector<torch::jit::IValue> inputs;
  inputs.push_back(input.cuda());  
  at::Tensor outputs = lat_module.forward(inputs).toTensor();
  hipStreamSynchronize(0);
  //cout<<outputs<<endl;
  double check4= wtime();
  inf+= (check4-check3);
  //cout<<"Inference done\n";
  H_ERR(hipMemcpy(output, outputs.data_ptr<float>(), sizeof(float) * Total_Trace*22, hipMemcpyHostToDevice));

  update<<<4096,64>>>(rob_d, output, factor_d, mean_d, status, Total_Trace);
  H_ERR(hipDeviceSynchronize());
  //cout<<"Update done\n";
  double check5=wtime();
  upd+=(check5-check4);
  iteration++;
}
fclose(pFile);
printf("%.4f, %.4f, %.4f, %.4f, %.4f\n",red, tr, pre, inf, upd);
double end_ = wtime();

gettimeofday(&total_end, NULL);
result<<<1, 1>>>(rob_d, Total_Trace, Instructions);
H_ERR(hipDeviceSynchronize());
double total_time = total_end.tv_sec - total_start.tv_sec + (total_end.tv_usec - total_start.tv_usec) / 1000000.0;
//cout << "Total time: " << (end_ - start_) << endl;
#ifdef RUN_TRUTH
cout << "Truth"
     << "\n";
#endif
return 0;
     cout << Instructions << " instructions finish by " << (curTick - 1) << "\n";
cout << "Time: " << total_time << "\n";
cout << "MIPS: " << Instructions / total_time / 1000000.0 << "\n";
cout << "USPI: " << total_time * 1000000.0 / Instructions << "\n";
cout << "Measured Time: " << measured_time / Instructions << "\n";
cout << "Cases: " << Case0 << " " << Case1 << " " << Case2 << " " << Case3 << " " << Case4 << " " << Case5 << "\n";
cout << "Trace: " << argv[1] << "\n";
#ifdef CLASSIFY
cout << "Model: " << argv[3] << " " << argv[4] << "\n";
#else
  //cout << "Lat Model: " << argv[3] << "\n";
#endif
return 0;
}
